// ===------ LapackUtils_api_test12.cu -------------------- *- CUDA -* ---=== //
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===--------------------------------------------------------------------=== //

// TEST_FEATURE: LapackUtils_potrf_scratchpad_size


#include <hip/hip_runtime.h>
#include "hipsolver.h"

int main() {
  float* a_s;
  hipsolverHandle_t handle;
  size_t device_ws_size_s;
  size_t host_ws_size_s;
  hipsolverDnParams_t params;

  hipsolverDnXpotrf_bufferSize(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, HIP_R_32F, a_s, 3, HIP_R_32F, &device_ws_size_s, &host_ws_size_s);
  return 0;
}
