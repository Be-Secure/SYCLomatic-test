// ====------------ math-ext-double.cu---------- *- CUDA -* -------------===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===---------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <vector>

using namespace std;

typedef vector<double> d_vector;
typedef pair<double, int> di_pair;

int passed = 0;
int failed = 0;

void check(bool IsPassed) {
  if (IsPassed) {
    cout << " ---- passed" << endl;
    passed++;
  } else {
    cout << " ---- failed" << endl;
    failed++;
  }
}

template <typename T = double>
void checkResult(const string &FuncName, const vector<T> &Inputs,
                 const double &Expect, const double &DeviceResult,
                 const int precision) {
  cout << FuncName << "(" << Inputs[0];
  for (size_t i = 1; i < Inputs.size(); ++i) {
    cout << ", " << Inputs[i];
  }
  cout << ") = " << fixed << setprecision(precision) << DeviceResult
       << " (expect " << Expect - pow(10, -precision) << " ~ "
       << Expect + pow(10, -precision) << ")";
  cout.unsetf(ios::fixed);
  check(abs(DeviceResult - Expect) < pow(10, -precision));
}

__global__ void _erfcinv(double *const DeviceResult, double Input) {
  *DeviceResult = erfcinv(Input);
}

void testErfcinv(double *const DeviceResult, double Input) {
  _erfcinv<<<1, 1>>>(DeviceResult, Input);
  hipDeviceSynchronize();
  // TODO: Need test host side.
}

void testErfcinvCases(const vector<pair<double, di_pair>> &TestCases) {
  double *DeviceResult;
  hipMallocManaged(&DeviceResult, sizeof(*DeviceResult));
  // Boundary values.
  testErfcinv(DeviceResult, 0);
  cout << "erfcinv(" << 0 << ") = " << *DeviceResult << " (expect inf)";
  check(*DeviceResult > 999999.9);
  testErfcinv(DeviceResult, 2);
  cout << "erfcinv(" << 2 << ") = " << *DeviceResult << " (expect -inf)";
  check(*DeviceResult < -999999.9);
  // Other test values.
  for (const auto &TestCase : TestCases) {
    testErfcinv(DeviceResult, TestCase.first);
    checkResult("erfcinv", {TestCase.first}, TestCase.second.first,
                *DeviceResult, TestCase.second.second);
  }
}

__global__ void _erfinv(double *const DeviceResult, double Input) {
  *DeviceResult = erfinv(Input);
}

void testErfinv(double *const DeviceResult, double Input) {
  _erfinv<<<1, 1>>>(DeviceResult, Input);
  hipDeviceSynchronize();
  // Call from host.
}

void testErfinvCases(const vector<pair<double, di_pair>> &TestCases) {
  double *DeviceResult;
  hipMallocManaged(&DeviceResult, sizeof(*DeviceResult));
  // Boundary values.
  testErfinv(DeviceResult, -1);
  cout << "erfinv(" << -1 << ") = " << *DeviceResult << " (expect -inf)";
  check(*DeviceResult < -999999.9);
  testErfinv(DeviceResult, 1);
  cout << "erfinv(" << 1 << ") = " << *DeviceResult << " (expect inf)";
  check(*DeviceResult > 999999.9);
  // Other test values.
  for (const auto &TestCase : TestCases) {
    testErfinv(DeviceResult, TestCase.first);
    checkResult("erfinv", {TestCase.first}, TestCase.second.first,
                *DeviceResult, TestCase.second.second);
  }
}

__global__ void setVecValue(double *Input1, const double Input2) {
  *Input1 = Input2;
}

__global__ void _norm(double *const DeviceResult, int Input1,
                      const double *Input2) {
  *DeviceResult = norm(Input1, Input2);
}

void testNorm(double *const DeviceResult, int Input1, const double *Input2) {
  _norm<<<1, 1>>>(DeviceResult, Input1, Input2);
  hipDeviceSynchronize();
  // Call from host.
}

void testNormCases(const vector<pair<d_vector, di_pair>> &TestCases) {
  double *DeviceResult;
  hipMallocManaged(&DeviceResult, sizeof(*DeviceResult));
  for (const auto &TestCase : TestCases) {
    double *Input;
    hipMallocManaged(&Input, TestCase.first.size() * sizeof(*Input));
    for (size_t i = 0; i < TestCase.first.size(); ++i) {
      // Notice: cannot set value from host!
      setVecValue<<<1, 1>>>(Input + i, TestCase.first[i]);
      hipDeviceSynchronize();
    }
    testNorm(DeviceResult, TestCase.first.size(), Input);
    string arg = "&{";
    for (size_t i = 0; i < TestCase.first.size() - 1; ++i) {
      arg += to_string(TestCase.first[i]) + ", ";
    }
    arg += to_string(TestCase.first.back()) + "}";
    checkResult<string>("norm", {to_string(TestCase.first.size()), arg},
                        TestCase.second.first, *DeviceResult,
                        TestCase.second.second);
  }
}

__global__ void _normcdf(double *const DeviceResult, double Input) {
  *DeviceResult = normcdf(Input);
}

void testNormcdf(double *const DeviceResult, double Input) {
  _normcdf<<<1, 1>>>(DeviceResult, Input);
  hipDeviceSynchronize();
  // Call from host.
}

void testNormcdfCases(const vector<pair<double, di_pair>> &TestCases) {
  double *DeviceResult;
  hipMallocManaged(&DeviceResult, sizeof(*DeviceResult));
  for (const auto &TestCase : TestCases) {
    testNormcdf(DeviceResult, TestCase.first);
    checkResult("normcdf", {TestCase.first}, TestCase.second.first,
                *DeviceResult, TestCase.second.second);
  }
}

__global__ void _normcdfinv(double *const DeviceResult, double Input) {
  *DeviceResult = normcdfinv(Input);
}

void testNormcdfinv(double *const DeviceResult, double Input) {
  _normcdfinv<<<1, 1>>>(DeviceResult, Input);
  hipDeviceSynchronize();
  // Call from host.
}

void testNormcdfinvCases(const vector<pair<double, di_pair>> &TestCases) {
  double *DeviceResult;
  hipMallocManaged(&DeviceResult, sizeof(*DeviceResult));
  // Boundary values.
  testNormcdfinv(DeviceResult, 0);
  cout << "normcdfinv(" << 0 << ") = " << *DeviceResult << " (expect -inf)";
  check(*DeviceResult < -999999.9);
  testNormcdfinv(DeviceResult, 1);
  cout << "normcdfinv(" << 1 << ") = " << *DeviceResult << " (expect inf)";
  check(*DeviceResult > 999999.9);
  // Other test values.
  for (const auto &TestCase : TestCases) {
    testNormcdfinv(DeviceResult, TestCase.first);
    checkResult("normcdfinv", {TestCase.first}, TestCase.second.first,
                *DeviceResult, TestCase.second.second);
  }
}

__global__ void _rnorm(double *const DeviceResult, int Input1,
                       const double *Input2) {
  *DeviceResult = rnorm(Input1, Input2);
}

void testRnorm(double *const DeviceResult, int Input1, const double *Input2) {
  _rnorm<<<1, 1>>>(DeviceResult, Input1, Input2);
  hipDeviceSynchronize();
  // Call from host.
}

void testRnormCases(const vector<pair<d_vector, di_pair>> &TestCases) {
  double *DeviceResult;
  hipMallocManaged(&DeviceResult, sizeof(*DeviceResult));
  for (const auto &TestCase : TestCases) {
    double *Input;
    hipMallocManaged(&Input, TestCase.first.size() * sizeof(*Input));
    for (size_t i = 0; i < TestCase.first.size(); ++i) {
      // Notice: cannot set value from host!
      setVecValue<<<1, 1>>>(Input + i, TestCase.first[i]);
      hipDeviceSynchronize();
    }
    testRnorm(DeviceResult, TestCase.first.size(), Input);
    string arg = "&{";
    for (size_t i = 0; i < TestCase.first.size() - 1; ++i) {
      arg += to_string(TestCase.first[i]) + ", ";
    }
    arg += to_string(TestCase.first.back()) + "}";
    checkResult<string>("rnorm", {to_string(TestCase.first.size()), arg},
                        TestCase.second.first, *DeviceResult,
                        TestCase.second.second);
  }
}

int main() {
  testErfcinvCases({
      {0.3, {0.732869077959217, 15}},
      {0.5, {0.4769362762044698, 16}},
      {0.8, {0.1791434546212916, 16}},
      {1.6, {-0.595116081449995, 15}},
  });
  testErfinvCases({
      {-0.3, {-0.2724627147267544, 16}},
      {-0.5, {-0.4769362762044698, 16}},
      {0, {0, 37}},
      {0.5, {0.4769362762044698, 16}},
  });
  testNormCases({
      {{-0.3, -0.34, -0.98}, {1.079814798935447, 15}},
      {{0.3, 0.34, 0.98}, {1.079814798935447, 15}},
      {{0.5}, {0.5, 16}},
      {{23, 432, 23, 456, 23}, {629.4020972319682, 13}},
  });
  testNormcdfCases({
      {-5, {0.0000002866515718791939, 22}},
      {-3, {0.00134989803163009458, 20}},
      {0, {0.5, 16}},
      {1, {0.841344746068543, 15}},
      {5, {0.9999997133484281, 16}},
  });
  testNormcdfinvCases({
      {0.3, {-0.524400512708041, 15}},
      {0.5, {0, 37}},
      {0.8, {0.841621233572915, 15}},
  });
  testRnormCases({
      {{-0.3, -0.34, -0.98}, {0.926084733220795, 15}},
      {{0.3, 0.34, 0.98}, {0.926084733220795, 15}},
      {{0.5}, {2, 16}},
      {{23, 432, 23, 456, 23}, {0.001588809450108087, 18}},
  });
  cout << "passed " << passed << "/" << passed + failed << " cases!" << endl;
  if (failed) {
    cout << "failed!" << endl;
  }
  return failed;
}
