// ====------------ math-emu-double.cu---------- *- CUDA -* -------------===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===---------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <vector>

using namespace std;

typedef vector<double> d_vector;
typedef tuple<double, double, double> d_tuple3;
typedef tuple<double, double, double, double> d_tuple4;
typedef pair<double, int> di_pair;

int passed = 0;
int failed = 0;

void check(bool IsPassed) {
  if (IsPassed) {
    cout << " ---- passed" << endl;
    passed++;
  } else {
    cout << " ---- failed" << endl;
    failed++;
  }
}

template <typename T = double>
void checkResult(const string &FuncName, const vector<T> &Inputs,
                 const double &Expect, const double &DeviceResult,
                 const int precision) {
  cout << FuncName << "(" << Inputs[0];
  for (size_t i = 1; i < Inputs.size(); ++i) {
    cout << ", " << Inputs[i];
  }
  cout << ") = " << fixed << setprecision(precision) << DeviceResult
       << " (expect " << Expect - pow(10, -precision) << " ~ "
       << Expect + pow(10, -precision) << ")";
  cout.unsetf(ios::fixed);
  check(abs(DeviceResult - Expect) < pow(10, -precision));
}

__global__ void setVecValue(double *Input1, const double Input2) {
  *Input1 = Input2;
}

__global__ void _norm(double *const DeviceResult, int Input1,
                      const double *Input2) {
  *DeviceResult = norm(Input1, Input2);
}

void testNorm(double *const DeviceResult, int Input1, const double *Input2) {
  _norm<<<1, 1>>>(DeviceResult, Input1, Input2);
  hipDeviceSynchronize();
  // TODO: Need test host side.
}

void testNormCases(const vector<pair<d_vector, di_pair>> &TestCases) {
  double *DeviceResult;
  hipMallocManaged(&DeviceResult, sizeof(*DeviceResult));
  // Other test values.
  for (const auto &TestCase : TestCases) {
    double *Input;
    hipMallocManaged(&Input, TestCase.first.size() * sizeof(*Input));
    for (size_t i = 0; i < TestCase.first.size(); ++i) {
      // Notice: cannot set value from host!
      setVecValue<<<1, 1>>>(Input + i, TestCase.first[i]);
      hipDeviceSynchronize();
    }
    testNorm(DeviceResult, TestCase.first.size(), Input);
    string arg = "&{";
    for (size_t i = 0; i < TestCase.first.size() - 1; ++i) {
      arg += to_string(TestCase.first[i]) + ", ";
    }
    arg += to_string(TestCase.first.back()) + "}";
    checkResult<string>("norm", {to_string(TestCase.first.size()), arg},
                        TestCase.second.first, *DeviceResult,
                        TestCase.second.second);
  }
}

__global__ void _norm3d(double *const DeviceResult, double Input1,
                        double Input2, double Input3) {
  *DeviceResult = norm3d(Input1, Input2, Input3);
}

void testNorm3d(double *const DeviceResult, double Input1, double Input2,
                double Input3) {
  _norm3d<<<1, 1>>>(DeviceResult, Input1, Input2, Input3);
  hipDeviceSynchronize();
  // Call from host.
}

void testNorm3dCases(const vector<pair<d_tuple3, di_pair>> &TestCases) {
  double *DeviceResult;
  hipMallocManaged(&DeviceResult, sizeof(*DeviceResult));
  for (const auto &TestCase : TestCases) {
    testNorm3d(DeviceResult, get<0>(TestCase.first), get<1>(TestCase.first),
               get<2>(TestCase.first));
    checkResult("norm3d",
                {get<0>(TestCase.first), get<1>(TestCase.first),
                 get<2>(TestCase.first)},
                TestCase.second.first, *DeviceResult, TestCase.second.second);
  }
}

__global__ void _norm4d(double *const DeviceResult, double Input1,
                        double Input2, double Input3, double Input4) {
  *DeviceResult = norm4d(Input1, Input2, Input3, Input4);
}

void testNorm4d(double *const DeviceResult, double Input1, double Input2,
                double Input3, double Input4) {
  _norm4d<<<1, 1>>>(DeviceResult, Input1, Input2, Input3, Input4);
  hipDeviceSynchronize();
  // Call from host.
}

void testNorm4dCases(const vector<pair<d_tuple4, di_pair>> &TestCases) {
  double *DeviceResult;
  hipMallocManaged(&DeviceResult, sizeof(*DeviceResult));
  for (const auto &TestCase : TestCases) {
    testNorm4d(DeviceResult, get<0>(TestCase.first), get<1>(TestCase.first),
               get<2>(TestCase.first), get<3>(TestCase.first));
    checkResult("norm4d",
                {get<0>(TestCase.first), get<1>(TestCase.first),
                 get<2>(TestCase.first), get<3>(TestCase.first)},
                TestCase.second.first, *DeviceResult, TestCase.second.second);
  }
}

__global__ void _normcdf(double *const DeviceResult, double Input) {
  *DeviceResult = normcdf(Input);
}

void testNormcdf(double *const DeviceResult, double Input) {
  _normcdf<<<1, 1>>>(DeviceResult, Input);
  hipDeviceSynchronize();
  // Call from host.
}

void testNormcdfCases(const vector<pair<double, di_pair>> &TestCases) {
  double *DeviceResult;
  hipMallocManaged(&DeviceResult, sizeof(*DeviceResult));
  // Other test values.
  for (const auto &TestCase : TestCases) {
    testNormcdf(DeviceResult, TestCase.first);
    checkResult("normcdf", {TestCase.first}, TestCase.second.first,
                *DeviceResult, TestCase.second.second);
  }
}

__global__ void _rnorm(double *const DeviceResult, int Input1,
                       const double *Input2) {
  *DeviceResult = rnorm(Input1, Input2);
}

void testRnorm(double *const DeviceResult, int Input1, const double *Input2) {
  _rnorm<<<1, 1>>>(DeviceResult, Input1, Input2);
  hipDeviceSynchronize();
  // Call from host.
}

void testRnormCases(const vector<pair<d_vector, di_pair>> &TestCases) {
  double *DeviceResult;
  hipMallocManaged(&DeviceResult, sizeof(*DeviceResult));
  // Other test values.
  for (const auto &TestCase : TestCases) {
    double *Input;
    hipMallocManaged(&Input, TestCase.first.size() * sizeof(*Input));
    for (size_t i = 0; i < TestCase.first.size(); ++i) {
      // Notice: cannot set value from host!
      setVecValue<<<1, 1>>>(Input + i, TestCase.first[i]);
      hipDeviceSynchronize();
    }
    testRnorm(DeviceResult, TestCase.first.size(), Input);
    string arg = "&{";
    for (size_t i = 0; i < TestCase.first.size() - 1; ++i) {
      arg += to_string(TestCase.first[i]) + ", ";
    }
    arg += to_string(TestCase.first.back()) + "}";
    checkResult<string>("rnorm", {to_string(TestCase.first.size()), arg},
                        TestCase.second.first, *DeviceResult,
                        TestCase.second.second);
  }
}

__global__ void _rnorm3d(double *const DeviceResult, double Input1,
                         double Input2, double Input3) {
  *DeviceResult = rnorm3d(Input1, Input2, Input3);
}

void testRnorm3d(double *const DeviceResult, double Input1, double Input2,
                 double Input3) {
  _rnorm3d<<<1, 1>>>(DeviceResult, Input1, Input2, Input3);
  hipDeviceSynchronize();
  // Call from host.
}

void testRnorm3dCases(const vector<pair<d_tuple3, di_pair>> &TestCases) {
  double *DeviceResult;
  hipMallocManaged(&DeviceResult, sizeof(*DeviceResult));
  for (const auto &TestCase : TestCases) {
    testRnorm3d(DeviceResult, get<0>(TestCase.first), get<1>(TestCase.first),
                get<2>(TestCase.first));
    checkResult("rnorm3d",
                {get<0>(TestCase.first), get<1>(TestCase.first),
                 get<2>(TestCase.first)},
                TestCase.second.first, *DeviceResult, TestCase.second.second);
  }
}

__global__ void _rnorm4d(double *const DeviceResult, double Input1,
                         double Input2, double Input3, double Input4) {
  *DeviceResult = rnorm4d(Input1, Input2, Input3, Input4);
}

void testRnorm4d(double *const DeviceResult, double Input1, double Input2,
                 double Input3, double Input4) {
  _rnorm4d<<<1, 1>>>(DeviceResult, Input1, Input2, Input3, Input4);
  hipDeviceSynchronize();
  // Call from host.
}

void testRnorm4dCases(const vector<pair<d_tuple4, di_pair>> &TestCases) {
  double *DeviceResult;
  hipMallocManaged(&DeviceResult, sizeof(*DeviceResult));
  for (const auto &TestCase : TestCases) {
    testRnorm4d(DeviceResult, get<0>(TestCase.first), get<1>(TestCase.first),
                get<2>(TestCase.first), get<3>(TestCase.first));
    checkResult("rnorm4d",
                {get<0>(TestCase.first), get<1>(TestCase.first),
                 get<2>(TestCase.first), get<3>(TestCase.first)},
                TestCase.second.first, *DeviceResult, TestCase.second.second);
  }
}

int main() {
  testNormCases({
      {{-0.3, -0.34, -0.98}, {1.079814798935447, 15}},
      {{0.3, 0.34, 0.98}, {1.079814798935447, 15}},
      {{0.5}, {0.5, 16}},
      {{23, 432, 23, 456, 23}, {629.4020972319682, 13}},
  });
  testNorm3dCases({
      {{-0.3, -0.34, -0.98}, {1.079814798935447, 15}},
      {{0.3, 0.34, 0.98}, {1.079814798935447, 15}},
      {{0.5, 456, 23}, {456.5799491874342, 13}},
      {{23, 432, 23}, {433.222806417206, 13}},
  });
  testNorm4dCases({
      {{-0.3, -0.34, -0.98, 1}, {1.471733671558818, 15}},
      {{0.3, 0.34, 0.98, 1}, {1.471733671558818, 15}},
      {{0.5, 456, 23, 1}, {456.5810442845827, 13}},
      {{23, 432, 23, 1}, {433.2239605562001, 13}},
  });
  testNormcdfCases({
      {-5, {0.000000286651571879194, 21}},
      {-3, {0.001349898031630095, 18}},
      {0, {0.5, 16}},
      {1, {0.841344746068543, 15}},
      {5, {0.9999997133484281, 16}},
  });
  testRnormCases({
      {{-0.3, -0.34, -0.98}, {0.926084733220795, 15}},
      {{0.3, 0.34, 0.98}, {0.926084733220795, 15}},
      {{0.5}, {2, 15}},
      {{23, 432, 23, 456, 23}, {0.001588809450108087, 18}},
  });
  testRnorm3dCases({
      {{-0.3, -0.34, -0.98}, {0.926084733220795, 15}},
      {{0.3, 0.34, 0.98}, {0.926084733220795, 15}},
      {{0.5, 456, 23}, {0.002190196923407782, 18}},
      {{23, 432, 23}, {0.002308281062740199, 18}},
  });
  testRnorm4dCases({
      {{-0.3, -0.34, -0.98, 1}, {0.679470762492529, 15}},
      {{0.3, 0.34, 0.98, 1}, {0.679470762492529, 15}},
      {{0.5, 456, 23, 1}, {0.002190191670280358, 18}},
      {{23, 432, 23, 1}, {0.002308274913317669, 18}},
  });
  cout << "passed " << passed << "/" << passed + failed << " cases!" << endl;
  if (failed) {
    cout << "failed!" << endl;
  }
  return failed;
}
